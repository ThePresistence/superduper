#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cerrno>
#include <cmath>
#include <time.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipsparse.h"


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////   Prototypes for BLAS and LAPACK functions.                                                                              ////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


extern "C"
{
  // Lapack routines.
  double dlamch_(const char   &cmach);

  void   dsyev_ (const char   &jobz,
                 const char   &uplo,
                 const int    &n,
                 double       *a,
                 const int    &lda,
                 double       *w,
                 double       *work,
                 int          &lwork,
                 int          &info);

  void   dsyevx_(const char   &jobz,
                 const char   &range,
                 const char   &uplo,
                 const int    &n,
                 double       *a,
                 const int    &lda,
                 const double &vl,
                 const double &vu,
                 const int    &il,
                 const int    &iu,
                 const double &abstol,
                 int          &m,
                 double       *w,
                 double       *z,
                 const int    &ldz,
                 double       *work,
                 const int    &lwork,
                 int          *iwork,
                 int          *ifail,
                 int          &info);

  // Prototype for private function in "gugatime.c".
  void gettime(double *tuser, double *tsys, double *twall);
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////   Device functions.                                                                                                      ////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void gpuInitBasisVectors(double    *B_d,
                                    int        ldb,
                                    const int *jrefconf_d,
                                    int        ni,
                                    int        nj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < ni  &&  j < nj)
  {
    if (i == jrefconf_d[j] - 1)  B_d[ldb * j + i] = 1.0;
    else                         B_d[ldb * j + i] = 0.0;
  }
}


__global__ void gpuAddBasisVectors(double *B_d,
                                   int     ldb,
                                   double *Q_d,
                                   int     ldq,
                                   double *Hii_d,
                                   double *eig_d,
                                   int    *iroot_d,
                                   double  qtol,
                                   int     ni,
                                   int     nj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < ni  &&  j < nj)
  {
    int    k         = iroot_d[j];
    double x         = eig_d[k] - Hii_d[i];

    // If x is too small, replace it by
    // +qtol or -qtol depending on its sign.
    //
    if (fabs(x) < qtol)
      x = copysign(qtol, x);

    B_d[ldb * j + i] = Q_d[ldq * k + i] / x;
  }
}


__global__ void gpuMultByDiag1(double       *A_d,
                               int           lda,
                               const double *b_d,
                               int           ni,
                               int           nj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < ni  &&  j < nj)
  {
    A_d[lda * j + i] *= b_d[j];
  }
}


__global__ void gpuMultByDiag2(double       *C_d,
                               int           ldc,
                               const double *A_d,
                               int           lda,
                               const double *b_d,
                               int           ni,
                               int           nj)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < ni  &&  j < nj)
  {
    C_d[ldc * j + i] = A_d[lda * j + i] * b_d[j];
  }
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////   Error handling functions.                                                                                              ////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


inline void Error(hipError_t err, const char *str)
{
  if (err != hipSuccess)
  {
    printf(" %s(): %s\n", str, hipGetErrorString(err));
    exit(1);
  }
}


inline void Error(hipblasStatus_t err, const char *str)
{
  if (err != HIPBLAS_STATUS_SUCCESS)
  {
    printf(" %s() returned error code %d.\n", str, err);
    exit(1);
  }
}


inline void Error(hipsparseStatus_t err, const char *str)
{
  if (err != HIPSPARSE_STATUS_SUCCESS)
  {
    printf(" %s() returned error code %d.\n", str, err);
    exit(1);
  }
}


inline void Sync(void)
{
  Error(hipDeviceSynchronize(), "hipDeviceSynchronize");
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////   Class definitions.                                                                                                     ////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// Namespace is necessary because gpudavliu.cu
// and cpudavliu.cpp share the same names.

namespace gpu
{

// Forward declarations.
struct DeviceMatBase;
struct DeviceTrans;
struct ColumnVec;
struct ScaledCol;
struct ProdMatVec;
struct ProdTraVec;
struct DeviceCSR;
struct HostMatBase;


struct BlaHandleWrapper
{
  hipblasHandle_t handle;

  BlaHandleWrapper(void)
  {
    Error(hipblasCreate(&handle), "hipblasCreate");
  }

 ~BlaHandleWrapper(void)
  {
    Error(hipblasDestroy(handle), "hipblasDestroy");
  }

  operator hipblasHandle_t (void)
  {
    return handle;
  }
};


struct SpaHandleWrapper
{
  hipsparseHandle_t handle;

  SpaHandleWrapper(void)
  {
    Error(hipsparseCreate(&handle), "hipsparseCreate");
  }

 ~SpaHandleWrapper(void)
  {
    Error(hipsparseDestroy(handle), "hipsparseDestroy");
  }

  operator hipsparseHandle_t (void)
  {
    return handle;
  }
};


struct MatDescrWrapper
{
  hipsparseMatDescr_t descrA;

  MatDescrWrapper(hipsparseMatrixType_t matrixType,
                  hipsparseFillMode_t   fillMode,
                  hipsparseDiagType_t   diagType,
                  hipsparseIndexBase_t  indexBase)
  {
    Error(hipsparseCreateMatDescr(&descrA),             "hipsparseCreateMatDescr");
    Error(hipsparseSetMatType     (descrA, matrixType), "hipsparseSetMatType");
    Error(hipsparseSetMatFillMode (descrA, fillMode),   "hipsparseSetMatFillMode");
    Error(hipsparseSetMatDiagType (descrA, diagType),   "hipsparseSetMatDiagType");
    Error(hipsparseSetMatIndexBase(descrA, indexBase),  "hipsparseSetMatIndexBase");
  }


 ~MatDescrWrapper(void)
  {
    Error(hipsparseDestroyMatDescr(descrA), "hipsparseDestroyMatDescr");
  }


  operator hipsparseMatDescr_t (void) const
  {
    return descrA;
  }
};


struct Counters
{
  long isync;
  long imemcpy;
  long idcopy;
  long idnrm2s;  // hipblasDnrm2(), ein Aufruf vor Sync().
  long idnrm2p;  // hipblasDnrm2(), mehrere Aufrufe vor Sync().
  long idscal;
  long idgemv;
  long idgemm;
  long idcsrmm;
  long ieigen;
  long iother;


  void Init(void)
  {
    isync   = 0L;
    imemcpy = 0L;
    idcopy  = 0L;
    idnrm2s = 0L;
    idnrm2p = 0L;
    idscal  = 0L;
    idgemv  = 0L;
    idgemm  = 0L;
    idcsrmm = 0L;
    ieigen  = 0L;
    iother  = 0L;
  }


  long Total(void)
  {
    return isync + imemcpy + idcopy + idnrm2s + idnrm2p + idscal + idgemv + idgemm + idcsrmm + ieigen + iother;
  };
};


struct Timing
{
  static Counters ncalls;
  static Counters twall;

  static void Init(void)
  {
    ncalls.Init();
    twall.Init();
  }


  static long GetTime(clockid_t clk_id)
  {
    timespec ts;

    if (clock_gettime(clk_id, &ts))
    {
      printf("clock_gettime(): %s\n", strerror(errno));
      exit(1);
    }

    return 1000000000L * ts.tv_sec + ts.tv_nsec;
  }


  static void Print(void)
  {
    long MiB = (ncalls.imemcpy + 524288L) / 1048576L;
    printf("\n Wall clock time statistics of function calls:\n");
    printf(" init:  %6ld calls,%12.4le s\n", ncalls.isync,   1e-9 * twall.isync);
    printf(" memcpy:%6ld MiB,  %12.4le s\n", MiB,            1e-9 * twall.imemcpy);
    printf(" dcopy: %6ld calls,%12.4le s\n", ncalls.idcopy,  1e-9 * twall.idcopy);
    printf(" dnrm2s:%6ld calls,%12.4le s\n", ncalls.idnrm2s, 1e-9 * twall.idnrm2s);
    printf(" dnrm2p:%6ld calls,%12.4le s\n", ncalls.idnrm2p, 1e-9 * twall.idnrm2p);
    printf(" dscal: %6ld calls,%12.4le s\n", ncalls.idscal,  1e-9 * twall.idscal);
    printf(" dgemv: %6ld calls,%12.4le s\n", ncalls.idgemv,  1e-9 * twall.idgemv);
    printf(" dgemm: %6ld calls,%12.4le s\n", ncalls.idgemm,  1e-9 * twall.idgemm);
    printf(" dcsrmm:%6ld calls,%12.4le s\n", ncalls.idcsrmm, 1e-9 * twall.idcsrmm);
    printf(" eigen: %6ld calls,%12.4le s\n", ncalls.ieigen,  1e-9 * twall.ieigen);
    printf(" other: %6ld calls,%12.4le s\n", ncalls.iother,  1e-9 * twall.iother);
    printf(" Total:            %14.4le s\n",                 1e-9 * twall.Total());
  }
};


template <class T> struct DeviceVecBase
{
  T      *v;
  size_t  n;

  DeviceVecBase(size_t m, T *b_h):  v(b_h), n(m)  {}
  DeviceVecBase(void)  {}
 ~DeviceVecBase(void)  {}


  operator T * (void) const
  {
    return v;
  }


  T &operator [] (int i) const
  {
    return v[i];
  }


  const DeviceVecBase &mult(const DeviceMatBase &A, const double *b, double alpha, double beta) const;
  const DeviceVecBase &mult(const DeviceTrans   &A, const double *b, double alpha, double beta) const;
  const DeviceVecBase &operator  = (const ProdMatVec &prod) const;
  const DeviceVecBase &operator  = (const ProdTraVec &prod) const;
  const DeviceVecBase &operator += (const ProdMatVec &prod) const;
  const DeviceVecBase &operator += (const ProdTraVec &prod) const;
  const DeviceVecBase &operator -= (const ProdMatVec &prod) const;
  const DeviceVecBase &operator -= (const ProdTraVec &prod) const;


  void copyFromHost(const T *b, size_t n) const
  {
    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipMemcpy(v, b, n * sizeof(T), hipMemcpyHostToDevice), "hipMemcpy");
    Timing::twall.imemcpy  += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.imemcpy += n * (long)sizeof(T);
  }
};


template <class T> struct DeviceVecMem: public DeviceVecBase<T>
{
  DeviceVecMem(size_t m, int prtlevel)
  {
    T           *tp;
    size_t       size  = m * sizeof(T);
    hipError_t  err   = hipMalloc(&tp, size);
    DeviceVecBase<T>::v = tp;
    DeviceVecBase<T>::n = m;

    if (err != hipSuccess)
    {
      printf(" Failed to allocate %ld bytes of global memory on CUDA device.\n", size);
      printf("hipMalloc(): %s\n", hipGetErrorString(err));
      exit(1);
    }

    if (prtlevel >= 2)
      printf(" Allocated %10ld bytes of GPU memory.\n", size);
  }


 ~DeviceVecMem(void)
  {
    Error(hipFree(DeviceVecBase<T>::v), "hipFree");
  }
};


template <class T> struct DeviceVec: public DeviceVecMem<T>
{
  DeviceVec(size_t m,               int prtlevel):  DeviceVecMem<T>(m, prtlevel)  {}


  DeviceVec(size_t m, const T *b_h, int prtlevel):  DeviceVecMem<T>(m, prtlevel)
  {
    this->copyFromHost(b_h, m);
  }


 ~DeviceVec(void)  {}
};


template <class T> struct HostVec
{
  T *v;

  HostVec(size_t n)
  {
    if (!(v = new T[n]))
    {
      printf(" Failed to allocate %ld bytes of CPU memory.\n", n * sizeof(T));
      exit(1);
    }
  }


 ~HostVec(void)
  {
    delete [] v;
  }


  operator T * (void) const
  {
    return v;
  }


  T &operator [] (int i) const
  {
    return v[i];
  }
};


struct ColumnVec
{
  const DeviceMatBase &M;
  int                  j;

  ColumnVec(const DeviceMatBase &M, int j);
 ~ColumnVec(void)  {}

  operator double * (void) const;

  const ColumnVec &mult(const DeviceMatBase &A, const double *b, double alpha, double beta) const;
  const ColumnVec &operator  = (const ScaledCol  &scol) const;
  const ColumnVec &operator  = (const ProdMatVec &prod) const;
  const ColumnVec &operator += (const ProdMatVec &prod) const;
  const ColumnVec &operator -= (const ProdMatVec &prod) const;
  const ColumnVec &operator *= (double b) const;
  const ScaledCol  operator *  (double b) const;
};


struct ScaledCol
{
  const ColumnVec &a;
  double           b;

  ScaledCol(const ColumnVec &a, double b):  a(a), b(b)  {}
 ~ScaledCol(void)  {}
};


struct ProdMatVec
{
  const DeviceMatBase &A;
  const double        *b;

  ProdMatVec(const DeviceMatBase &A, const double *b):  A(A), b(b)  {}
 ~ProdMatVec(void)  {}
};


struct ProdTraVec
{
  const DeviceTrans &A;
  const double      *b;

  ProdTraVec(const DeviceTrans &A, const double *b):  A(A), b(b)  {}
 ~ProdTraVec(void)  {}
};


struct ProdMatMat
{
  const DeviceMatBase &A;
  const DeviceMatBase &B;

  ProdMatMat(const DeviceMatBase &A, const DeviceMatBase &B):  A(A), B(B)  {}
 ~ProdMatMat(void)  {}
};


struct ProdTraMat
{
  const DeviceTrans   &A;
  const DeviceMatBase &B;

  ProdTraMat(const DeviceTrans &A, const DeviceMatBase &B):  A(A), B(B)  {}
 ~ProdTraMat(void)  {}
};


struct ProdSpaDen
{
  const DeviceCSR     &A;
  const DeviceMatBase &B;

  ProdSpaDen(const DeviceCSR &A, const DeviceMatBase &B):  A(A), B(B)  {}
 ~ProdSpaDen(void)  {}
};


struct DeviceTrans
{
  hipblasHandle_t  handle;
  double         *M;
  int             ld;
  int             ni;  // Number of rows of the transposed matrix.
  int             nj;  // Number of columns of the transposed matrix.

  DeviceTrans(hipblasHandle_t h, double *M, int ld, int mi, int mj):  handle(h), M(M), ld(ld), ni(mi), nj(mj)  {}

  operator double * (void) const
  {
    return M;
  }

  ProdTraVec operator * (const ColumnVec             &b) const;
  ProdTraVec operator * (const DeviceVecBase<double> &b) const;
  ProdTraMat operator * (const DeviceMatBase         &B) const;
};


struct DeviceSym
{
  double *M;
  int     ld;
  int     ni;  // Number of rows (and columns) of the symmetric matrix.

  DeviceSym(double *M, int ld, int mi):  M(M), ld(ld), ni(mi)  {}
  DeviceSym(int mi):                     ni(mi)                {}

  const DeviceSym &operator = (const ProdTraMat &prod) const;
};


struct DeviceCSR
{
  hipsparseHandle_t  handle;
  int               nrows;
  int               nnz;
  DeviceVec<double> a;
  DeviceVec<int>    ia;
  DeviceVec<int>    ja;

  DeviceCSR(hipsparseHandle_t h, int n, const double *Hij, const int *ifirst, const int *icol, int prtlevel):
    handle(h), nrows(n), nnz(ifirst[n]-1), a(nnz, Hij, prtlevel), ia(n+1, ifirst, prtlevel), ja(nnz, icol, prtlevel)  {}

 ~DeviceCSR(void)  {}

  ProdSpaDen operator * (const DeviceMatBase &B) const
  {
    return ProdSpaDen(*this, B);
  }
};


struct DeviceMatBase
{
  hipblasHandle_t  handle;
  double         *M;
  int             ld;
  int             ni;
  int             nj;

  DeviceMatBase(hipblasHandle_t h, double *M, int ld, int mi, int mj):  handle(h), M(M), ld(ld), ni(mi), nj(mj)  {}
  DeviceMatBase(hipblasHandle_t h, int mi, int mj):                     handle(h), ni(mi), nj(mj)                {}


  operator double * (void) const
  {
    return M;
  }


  ColumnVec colvec(int j) const
  {
    return ColumnVec(*this, j);
  }


  DeviceMatBase submat(int mi, int mj) const
  {
    if (mi > ni  ||  mj > nj)
    {
      printf(" Submatrix too large.\n");
      exit(1);
    }

    return DeviceMatBase(handle, M, ld, mi, mj);
  }


  DeviceMatBase submat(int mi, int mj, int i, int j) const
  {
    if (mi + i > ni  ||  mj + j > nj)
    {
      printf(" Submatrix too large.\n");
      exit(1);
    }

    return DeviceMatBase(handle, M+ld*j+i, ld, mi, mj);
  }


  DeviceSym subsym(int mi) const
  {
    if (mi > ni  ||  mi > nj)
    {
      printf(" Submatrix too large.\n");
      exit(1);
    }

    return DeviceSym(M, ld, mi);
  }


  DeviceSym subsym(int mi, int i) const
  {
    if (mi + i > ni  ||  mi + i > nj)
    {
      printf(" Submatrix too large.\n");
      exit(1);
    }

    return DeviceSym(M+(ld+1)*i, ld, mi);
  }


  double &operator () (int i, int j) const
  {
    if (i >= ni  ||  j >= nj)
    {
      printf(" Subscript out of range.\n");
      exit(1);
    }

    return M[ld*j+i];
  }


  DeviceTrans operator ~ (void) const
  {
    return DeviceTrans(handle, M, ld, nj, ni);
  }


  ProdMatVec operator * (const ColumnVec &b) const
  {
    if (nj != b.M.ni)
    {
      printf(" Incompatible matrix dimensions during M * v.\n");
      printf("   M: %6d x %6d\n", ni, nj);
      printf("   v:       %6d\n", b.M.ni);
      exit(1);
    }

    return ProdMatVec(*this, b);
  }


  ProdMatVec operator * (const DeviceVecBase<double> &b) const
  {
    return ProdMatVec(*this, b);
  }


  ProdMatMat operator * (const DeviceMatBase &B) const
  {
    return ProdMatMat(*this, B);
  }


  const DeviceMatBase &mult(const DeviceMatBase &A, const DeviceMatBase &B, double alpha, double beta) const
  {
    if (ni != A.ni  ||  nj != B.nj  ||  A.nj != B.ni)
    {
      printf(" Incompatible matrix dimensions during C = A * B.\n");
      printf("   A: %6d x %6d\n", A.ni, A.nj);
      printf("   B: %6d x %6d\n", B.ni, B.nj);
      printf("   C: %6d x %6d\n", ni,   nj);
      exit(1);
    }

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, ni, nj, A.nj,
                      &alpha, A, A.ld, B, B.ld, &beta, M, ld), "hipblasDgemm");
    Sync();
    Timing::twall.idgemm += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.idgemm++;
    return *this;
  }


  const DeviceMatBase &mult(const DeviceTrans &A, const DeviceMatBase &B, double alpha, double beta) const
  {
    if (ni != A.ni  ||  nj != B.nj  ||  A.nj != B.ni)
    {
      printf(" Incompatible matrix dimensions during C = A**T * B.\n");
      printf("   A**T: %6d x %6d\n", A.ni, A.nj);
      printf("   B:    %6d x %6d\n", B.ni, B.nj);
      printf("   C:    %6d x %6d\n", ni,   nj);
      exit(1);
    }

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni, nj, A.nj,
                      &alpha, A, A.ld, B, B.ld, &beta, M, ld), "hipblasDgemm");
    Sync();
    Timing::twall.idgemm += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.idgemm++;
    return *this;
  }


  const DeviceMatBase &operator = (const ProdMatMat &prod) const
  {
    return mult(prod.A, prod.B, 1.0, 0.0);
  }


  const DeviceMatBase &operator = (const ProdTraMat &prod) const
  {
    return mult(prod.A, prod.B, 1.0, 0.0);
  }


  const DeviceMatBase &operator = (const ProdSpaDen &prod) const
  {
    if (ni != prod.A.nrows  ||  nj != prod.B.nj  ||  prod.A.nrows != prod.B.ni)
    {
      printf(" Incompatible matrix dimensions during C = CSR * B.\n");
      printf("   CSR: %6d x %6d\n", prod.A.nrows, prod.A.nrows);
      printf("   B:   %6d x %6d\n", prod.B.ni,    prod.B.nj);
      printf("   C:   %6d x %6d\n", ni,           nj);
      exit(1);
    }

    MatDescrWrapper descrA(HIPSPARSE_MATRIX_TYPE_SYMMETRIC,
                           HIPSPARSE_FILL_MODE_UPPER,
                           HIPSPARSE_DIAG_TYPE_NON_UNIT,
                           HIPSPARSE_INDEX_BASE_ONE);
    double alpha = 1.0;
    double beta  = 0.0;

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipsparseDcsrmm(prod.A.handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                         prod.A.nrows, prod.B.nj, prod.A.nrows, prod.A.nnz,
                         &alpha, descrA, prod.A.a, prod.A.ia, prod.A.ja,
                         prod.B, prod.B.ld, &beta, *this, ld), "hipsparseDcsrmm");
    Sync();
    Timing::twall.idcsrmm += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.idcsrmm++;
    return *this;
  }


  const DeviceMatBase &operator += (const ProdMatMat &prod) const
  {
    return mult(prod.A, prod.B, 1.0, 1.0);
  }


  const DeviceMatBase &operator += (const ProdTraMat &prod) const
  {
    return mult(prod.A, prod.B, 1.0, 1.0);
  }


  const DeviceMatBase &operator -= (const ProdMatMat &prod) const
  {
    return mult(prod.A, prod.B, -1.0, 1.0);
  }


  const DeviceMatBase &operator -= (const ProdTraMat &prod) const
  {
    return mult(prod.A, prod.B, -1.0, 1.0);
  }


  const DeviceMatBase &multByDiag(double *b_d) const
  {
    const dim3 blockDim(256, 1);
    const dim3 gridDim((ni + blockDim.x - 1) / blockDim.x, (nj + blockDim.y - 1) / blockDim.y);

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    gpuMultByDiag1<<<gridDim, blockDim>>>(M, ld, b_d, ni, nj);
    Sync();
    Timing::twall.iother += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.iother++;

    return *this;
  }


  const DeviceMatBase &multByDiag(const DeviceMatBase &A, double *b_d) const
  {
    const dim3 blockDim(256, 1);
    const dim3 gridDim((ni + blockDim.x - 1) / blockDim.x, (nj + blockDim.y - 1) / blockDim.y);

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    gpuMultByDiag2<<<gridDim, blockDim>>>(M, ld, A, A.ld, b_d, ni, nj);
    Sync();
    Timing::twall.iother += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.iother++;

    return *this;
  }


  void colnorms(double *b) const
  {
    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    for (int j=0; j<nj; ++j)
      Error(hipblasDnrm2(handle, ni, M+ld*j, 1, b+j), "hipblasDnrm2");
    Sync();
    Timing::twall.idnrm2p  += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.idnrm2p += nj;
  }


  void colnorm(int j, double &result) const
  {
    if (j >= nj)
    {
      printf(" Column index out of range.\n");
      exit(1);
    }

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipblasDnrm2(handle, ni, M+ld*j, 1, &result), "hipblasDnrm2");
    Sync();
    Timing::twall.idnrm2s += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.idnrm2s++;
  }


  void swap(DeviceMatBase &B)
  {
    // Swap handle.
    hipblasHandle_t h = B.handle;
    B.handle         = handle;
    handle           = h;

    // Swap pointer.
    double        *x = B.M;
    B.M              = M;
    M                = x;

    // Swap leading dimension.
    int            i = B.ld;
    B.ld             = ld;
    ld               = i;

    // Swap number of rows.
    i                = B.ni;
    B.ni             = ni;
    ni               = i;

    // Swap number of columns.
    i                = B.nj;
    B.nj             = nj;
    nj               = i;
  }


  void init(const int *jrefconf_d) const
  {
    const dim3 blockDim(256, 1);
    const dim3 gridDim((ni + blockDim.x - 1) / blockDim.x, (nj + blockDim.y - 1) / blockDim.y);

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    gpuInitBasisVectors<<<gridDim, blockDim>>>(M, ld, jrefconf_d, ni, nj);
    Sync();
    Timing::twall.iother += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.iother++;
  }


  void copyToHost(double *B, int ldb) const
  {
    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipMemcpy2D(B, ldb*sizeof(double),
                       M, ld *sizeof(double),
                       ni*sizeof(double), nj,
                       hipMemcpyDeviceToHost), "hipMemcpy2D");
    Timing::twall.imemcpy  += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.imemcpy += (long)ni * (long)nj * (long)sizeof(double);
  }


  void copyFromHost(const double *B, int ldb) const
  {
    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipMemcpy2D(M, ld *sizeof(double),
                       B, ldb*sizeof(double),
                       ni*sizeof(double), nj,
                       hipMemcpyHostToDevice), "hipMemcpy2D");
    Timing::twall.imemcpy  += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.imemcpy += (long)ni * (long)nj * (long)sizeof(double);
  }


  void copyFromHost(const HostMatBase &B) const;
  void diag(HostVec<double> &eig_h) const;
};


struct DeviceMat: public DeviceMatBase
{
  DeviceMat(hipblasHandle_t h, int ni, int nj, int prtlevel):  DeviceMatBase(h, ni, nj)
  {
    ld               = ((ni + 15) / 16) * 16;
    size_t      size = ld * nj * sizeof(double);
    hipError_t err  = hipMalloc(&M, size);

    if (err != hipSuccess)
    {
      printf(" Failed to allocate %ld bytes of global memory on CUDA device.\n", size);
      printf(" hipMalloc(): %s\n", hipGetErrorString(err));
      exit(1);
    }

    if (prtlevel >= 2)
      printf(" Allocated %10ld bytes of GPU memory.\n", size);
  }

 ~DeviceMat(void)
  {
    Error(hipFree(M), "hipFree");
  }
};


struct HostMatBase
{
  double *M;
  int     ld;
  int     ni;
  int     nj;

  HostMatBase(double *M, int ld, int mi, int mj):  M(M), ld(ld), ni(mi), nj(mj)  {}
  HostMatBase(int mi, int mj):                     ni(mi), nj(mj)                {}


  operator double * (void) const
  {
    return M;
  }


  double &operator () (int i, int j) const
  {
    return M[j*ld+i];
  }


  HostMatBase submat(int mi, int mj) const
  {
    if (mi > ni  ||  mj > nj)
    {
      printf(" Submatrix too large.\n");
      exit(1);
    }

    return HostMatBase(M, ld, mi, mj);
  }


  void copyFromDevice(DeviceMatBase &B) const
  {
    if (ni > B.ni  ||  nj > B.nj)
    {
      printf(" Incompatible matrix dimensions during copying.\n");
      exit(1);
    }

    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    Error(hipMemcpy2D(M, ld  *sizeof(double),
                       B, B.ld*sizeof(double),
                       ni*sizeof(double), nj,
                       hipMemcpyDeviceToHost), "hipMemcpy2D");
    Timing::twall.imemcpy  += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.imemcpy += (long)ni * (long)nj * (long)sizeof(double);
  }


  void diag(const HostVec<double> &eig_h) const
  {
    if (ni != nj)
    {
      printf(" Square matrix expected.\n");
      exit(1);
    }

    int    lwork = -1;
    int    info  =  0;
    double work1;
    dsyev_('V', 'L', ni, M, ld, eig_h, &work1, lwork, info);

    if (info != 0)
    {
      printf(" Error from DSYEV() when querying workspace size.\n");
      exit(1);
    }

    lwork = work1;
    HostVec<double> work(lwork);
    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    dsyev_('V', 'L', ni, M, ld, eig_h, work, lwork, info);
    Timing::twall.ieigen += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.ieigen++;

    if (info != 0)
    {
      printf(" Error from DSYEV() when diagonalizing.\n");
      exit(1);
    }
  }


  void eigen(const HostMatBase &U_h, const HostVec<double> &eig_h) const
  {
    if (ni != nj)
    {
      printf(" Square matrix expected.\n");
      exit(1);
    }

    if (U_h.ni != ni  ||  U_h.nj > ni)
    {
      printf(" Incompatible matrix dimensions.\n");
      exit(1);
    }

    HostVec<int> iwork(ni * 5);
    HostVec<int> ifail(ni);
    double       abstol = 2.0 * dlamch_('S');
    double       work1;
    int          lwork  = -1;
    int          info   =  0;
    int          m;

    dsyevx_('V', 'I', 'L', ni, M, ld, 0.0, 0.0, 1, U_h.nj, abstol, m, eig_h, U_h, U_h.ld, &work1, lwork, iwork, ifail, info);

    if (info != 0)
    {
      printf(" Error from DSYEVX() when querying workspace size.\n");
      exit(1);
    }

    lwork = work1;
    HostVec<double> work(lwork);
    long t1 = Timing::GetTime(CLOCK_MONOTONIC);
    dsyevx_('V', 'I', 'L', ni, M, ld, 0.0, 0.0, 1, U_h.nj, abstol, m, eig_h, U_h, U_h.ld, work, lwork, iwork, ifail, info);
    Timing::twall.ieigen += Timing::GetTime(CLOCK_MONOTONIC) - t1;
    Timing::ncalls.ieigen++;

    if (info != 0  ||  m < U_h.nj)
    {
      printf(" Error from DSYEVX() when diagonalizing.\n");
      exit(1);
    }
  }


  void Print(void)
  {
    for (int k=0; k<nj; k+=6)
    {
      for (int j=k; j<k+6; ++j)
	printf("%20d", j+1);

      printf("\n");

      for (int i=k; i<ni; ++i)
      {
	int l = k+6;
	if (nj  < l)  l = nj;
        if (i+1 < l)  l = i+1;

	printf("%8d", i+1);

	for (int j=k; j<l; ++j)
	  printf("%20.15lf", (*this)(i,j));

	printf("\n");
      }

      printf("\n");
    }
  }
};


struct HostMat: public HostMatBase
{
  HostMat(int ni, int nj):  HostMatBase(ni, nj)
  {
    ld           = ni;
    size_t nelem = ld * nj;

    if (!(M = new double[nelem]))
    {
      printf(" Failed to allocate %ld bytes of CPU memory.\n", nelem * sizeof(double));
      exit(1);
    }
  }

 ~HostMat(void)
  {
    delete [] M;
  }
};


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////   Member and friend functions.                                                                                           ////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


template <> inline const DeviceVecBase<double> &
  DeviceVecBase<double>::mult(const DeviceMatBase &A, const double *b, double alpha, double beta) const
{
  long t1 = Timing::GetTime(CLOCK_MONOTONIC);
  Error(hipblasDgemv(A.handle, HIPBLAS_OP_N, A.ni, A.nj, &alpha, A, A.ld, b, 1, &beta, *this, 1), "hipblasDgemv");
  Sync();
  Timing::twall.idgemv += Timing::GetTime(CLOCK_MONOTONIC) - t1;
  Timing::ncalls.idgemv++;
  return *this;
}


template <> inline const DeviceVecBase<double> &
  DeviceVecBase<double>::mult(const DeviceTrans &A, const double *b, double alpha, double beta) const
{
  long t1 = Timing::GetTime(CLOCK_MONOTONIC);
  Error(hipblasDgemv(A.handle, HIPBLAS_OP_T, A.nj, A.ni, &alpha, A, A.ld, b, 1, &beta, *this, 1), "hipblasDgemv");
  Sync();
  Timing::twall.idgemv += Timing::GetTime(CLOCK_MONOTONIC) - t1;
  Timing::ncalls.idgemv++;
  return *this;
}


template <> inline const DeviceVecBase<double> &DeviceVecBase<double>::operator = (const ProdMatVec &prod) const
{
  return mult(prod.A, prod.b, 1.0, 0.0);
}


template <> inline const DeviceVecBase<double> &DeviceVecBase<double>::operator = (const ProdTraVec &prod) const
{
  return mult(prod.A, prod.b, 1.0, 0.0);
}


template <> inline const DeviceVecBase<double> &DeviceVecBase<double>::operator += (const ProdMatVec &prod) const
{
  return mult(prod.A, prod.b, 1.0, 1.0);
}


template <> inline const DeviceVecBase<double> &DeviceVecBase<double>::operator += (const ProdTraVec &prod) const
{
  return mult(prod.A, prod.b, 1.0, 1.0);
}


template <> inline const DeviceVecBase<double> &DeviceVecBase<double>::operator -= (const ProdMatVec &prod) const
{
  return mult(prod.A, prod.b, -1.0, 1.0);
}


template <> inline const DeviceVecBase<double> &DeviceVecBase<double>::operator -= (const ProdTraVec &prod) const
{
  return mult(prod.A, prod.b, -1.0, 1.0);
}


inline ColumnVec::ColumnVec(const DeviceMatBase &M, int j):  M(M), j(j)
{
  if (j >= M.nj)
  {
    printf(" Column index out of range.");
    exit(1);
  }
}


inline ColumnVec::operator double * (void) const
{
  return M.M + M.ld * j;
}


inline const ColumnVec &ColumnVec::mult(const DeviceMatBase &A, const double *b, double alpha, double beta) const
{
  long t1 = Timing::GetTime(CLOCK_MONOTONIC);
  Error(hipblasDgemv(A.handle, HIPBLAS_OP_N, A.ni, A.nj, &alpha, A, A.ld, b, 1, &beta, *this, 1), "hipblasDgemv");
  Sync();
  Timing::twall.idgemv += Timing::GetTime(CLOCK_MONOTONIC) - t1;
  Timing::ncalls.idgemv++;
  return *this;
}


inline const ColumnVec &ColumnVec::operator = (const ScaledCol &scol) const
{
  if (M.ni != scol.a.M.ni)
  {
    printf(" Incompatible matrix dimensions during y = x * a.");
    exit(1);
  }

  // Copy and scale in two steps, there seems to be no combined function.
  long t1 = Timing::GetTime(CLOCK_MONOTONIC);
  Error(hipblasDcopy(M.handle, M.ni, scol.a, 1, *this, 1), "hipblasDcopy");
  Sync();
  long t2 = Timing::GetTime(CLOCK_MONOTONIC);
  Error(hipblasDscal(M.handle, M.ni, &scol.b, *this, 1), "hipblasDaxpy");
  Sync();
  Timing::twall.idscal += Timing::GetTime(CLOCK_MONOTONIC) - t2;
  Timing::twall.idcopy += t2 - t1;
  Timing::ncalls.idcopy++;
  Timing::ncalls.idscal++;
  return *this;
}


inline const ColumnVec &ColumnVec::operator = (const ProdMatVec &prod) const
{
  if (M.ni != prod.A.ni)
  {
    printf(" Incompatible matrix dimensions during c = A * b.");
    exit(1);
  }

  return mult(prod.A, prod.b, 1.0, 0.0);
}


inline const ColumnVec &ColumnVec::operator += (const ProdMatVec &prod) const
{
  if (M.ni != prod.A.ni)
  {
    printf(" Incompatible matrix dimensions during c = A * b.");
    exit(1);
  }

  return mult(prod.A, prod.b, 1.0, 1.0);
}


inline const ColumnVec &ColumnVec::operator -= (const ProdMatVec &prod) const
{
  if (M.ni != prod.A.ni)
  {
    printf(" Incompatible matrix dimensions during c = A * b.");
    exit(1);
  }

  return mult(prod.A, prod.b, -1.0, 1.0);
}


inline const ColumnVec &ColumnVec::operator *= (double b) const
{
  long t1 = Timing::GetTime(CLOCK_MONOTONIC);
  Error(hipblasDscal(M.handle, M.ni, &b, M, 1), "hipblasDscal");
  Sync();
  Timing::twall.idscal += Timing::GetTime(CLOCK_MONOTONIC) - t1;
  Timing::ncalls.idscal++;
  return *this;
}


const ScaledCol ColumnVec::operator * (double b) const
{
  return ScaledCol(*this, b);
}


inline ProdTraVec DeviceTrans::operator * (const ColumnVec &b) const
{
  if (nj != b.M.ni)
  {
    printf(" Incompatible matrix dimensions during M**T * v.\n");
    printf("   M**T: %6d x %6d\n", ni, nj);
    printf("   v:          %6d\n", b.M.ni);
    exit(1);
  }

  return ProdTraVec(*this, b);
}


inline ProdTraVec DeviceTrans::operator * (const DeviceVecBase<double> &b) const
{
  return ProdTraVec(*this, b);
}


inline ProdTraMat DeviceTrans::operator * (const DeviceMatBase &B) const
{
  return ProdTraMat(*this, B);
}


const DeviceSym &DeviceSym::operator = (const ProdTraMat &prod) const
{
  if (ni != prod.A.ni  ||  ni != prod.B.nj  ||  prod.A.nj != prod.B.ni)
  {
    printf(" Incompatible matrix dimensions during Sym = A**T * B.\n");
    printf("   A**T: %6d x %6d\n", prod.A.ni, prod.A.nj);
    printf("   B:    %6d x %6d\n", prod.B.ni, prod.B.nj);
    printf("   Sym:  %6d x %6d\n", ni,        ni);
    exit(1);
  }

  double alpha = 1.0;
  double beta  = 0.0;
  /*
  Error(hipblasDtrmm(prod.B.handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, HIPBLAS_DIAG_NON_UNIT,
                    prod.B.ni, prod.B.nj, &alpha, prod.A, prod.A.ld, prod.B, prod.B.ld, M, ld), "hipblasDtrmm");
  */
  long t1 = Timing::GetTime(CLOCK_MONOTONIC);
  Error(hipblasDgemm(prod.B.handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni, ni, prod.B.ni,
                    &alpha, prod.A, prod.A.ld, prod.B, prod.B.ld, &beta, M, ld), "hipblasDgemm");
  Sync();
  Timing::twall.idgemm += Timing::GetTime(CLOCK_MONOTONIC) - t1;
  Timing::ncalls.idgemm++;
  return *this;
}


inline void DeviceMatBase::copyFromHost(const HostMatBase &B) const
{
  if (ni > B.ni  ||  nj > B.nj)
  {
    printf(" Incompatible matrix dimensions during copying.\n");
    exit(1);
  }

  copyFromHost(B, B.ld);
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////   Host functions.                                                                                                        ////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


void AddBasisVectors(const DeviceMatBase     &B_d,
                     const DeviceMatBase     &Q_d,
                     const DeviceVec<double> &Hii_d,
                     const DeviceVec<double> &eig_d,
                     const DeviceVec<int>    &iroot_d,
                     double                   qtol)
{
  // Effectively, the following operation is performed:
  //
  // for (int i=0; i<B_d.ni; ++i)
  //   for (int j=0; j<B_d.nj; ++j)
  //   {
  //     int k    = iroot_d[j];
  //     double x = eig_d[k] - Hii[i];
  //
  //     if (fabs(x) < qtol)
  //       x = copysign(qtol, x);
  //
  //     B_d(i,j) = Q_d(i,k) / x;
  //   }

  const dim3 blockDim(256, 1);
  const dim3 gridDim((B_d.ni + blockDim.x - 1) / blockDim.x, (B_d.nj + blockDim.y - 1) / blockDim.y);

  long t1 = Timing::GetTime(CLOCK_MONOTONIC);
  gpuAddBasisVectors<<<gridDim, blockDim>>>(B_d, B_d.ld, Q_d, Q_d.ld, Hii_d, eig_d, iroot_d, qtol, B_d.ni, B_d.nj);
  Sync();
  Timing::twall.iother += Timing::GetTime(CLOCK_MONOTONIC) - t1;
  Timing::ncalls.iother++;
}


void Schmidt1(const DeviceMatBase &B_d,
              const DeviceMatBase &Q_d,
              const DeviceMatBase &A_d)
{
  // Schmidt orthogonalize the column vectors of Q_d against
  // the column vectors of B_d. A_d is used as scratch space.

  A_d  = ~B_d * Q_d;
  Q_d -=  B_d * A_d;
}


void Schmidt2(const DeviceMatBase         &B_d,
              const DeviceMatBase         &Q_d,
              const DeviceVecBase<double> &v_d,
              int                         &newdav,
              double                       qtol)
{
  // Schmidt orthogonalize the column vectors of Q among each other.
  // Vectors with a norm less than qtol after projecting out the
  // previous vectors are dropped. The final vectors are copied
  // into B. Their number is returned in newdav.

  int nci = B_d.ni;

  if (nci != Q_d.ni  ||  B_d.nj < Q_d.nj)
  {
    printf(" Incompatible matrix dimensions in Schmidt2().\n");
    printf("   B: %6d x %6d\n", B_d.ni, B_d.nj);
    printf("   Q: %6d x %6d\n", Q_d.ni, Q_d.nj);
    exit(1);
  }

  newdav = 0;

  for (int j=0; j<Q_d.nj; ++j)
  {
    if (newdav > 0)
    {
      // Project previous vectors out of current (j-th) vector.
      v_d            = ~B_d.submat(nci, newdav) * Q_d.colvec(j);
      Q_d.colvec(j) -=  B_d.submat(nci, newdav) * v_d;
    }

    double scalf;
    Q_d.colnorm(j, scalf);

    if (scalf > qtol)
    {
      // Normalize current vector and copy into B.
      scalf                = 1.0 / scalf;
      B_d.colvec(newdav++) = Q_d.colvec(j) * scalf;
    }
  }
}


void Schmidt3(const DeviceMatBase         &Q_d,
              const DeviceVecBase<double> &v_d)
{
  for (int j=0; j<Q_d.nj; ++j)
  {
    if (j > 0)
    {
      // Project previous vectors out of current (j-th) vector.
      v_d            = ~Q_d.submat(Q_d.ni, j) * Q_d.colvec(j);
      Q_d.colvec(j) -=  Q_d.submat(Q_d.ni, j) * v_d;
    }

    // Normalize current vector.
    double scalf;
    Q_d.colnorm(j, scalf);
    scalf          = 1.0 / scalf;
    Q_d.colvec(j) *= scalf;
  }
}


void DavLiu(double *Hii,
            double *Hij,
            int    *ifirst,
            int    *icol,
            int     nci,
            int    *jrefconf,
            int     inidav,
            double *E,
            double *C,
            int     ldc,
            int     nroots,
            int     mindav,
            int     maxdav,
            int     kitdav,
            double  qtol,
            int     prtlevel,
            int    &icall)
{
  Timing::Init();
  long              ibegin = Timing::GetTime(CLOCK_MONOTONIC);
  Sync();
  Timing::twall.isync      = Timing::GetTime(CLOCK_MONOTONIC) - ibegin;
  Timing::ncalls.isync++;
  BlaHandleWrapper  bla;
  SpaHandleWrapper  spa;
  DeviceVec<double> Hii_d(nci, Hii, prtlevel);
  DeviceCSR         Ham_d(spa, nci, Hij, ifirst, icol, prtlevel);
  DeviceMat         B_d(bla, nci, maxdav, prtlevel);
  DeviceMat         P_d(bla, nci, maxdav, prtlevel);
  DeviceMat         G_d(bla, maxdav, maxdav, prtlevel);
  DeviceMat         A_d(bla, maxdav, maxdav, prtlevel);
  HostMat           A_h(maxdav, maxdav);
  DeviceMat         U_d(bla, maxdav, maxdav, prtlevel);
  HostMat           U_h(maxdav, mindav);
  DeviceVec<double> v_d(maxdav, prtlevel);
  DeviceVec<double> eig_d(maxdav, prtlevel);
  HostVec<double>   eig_h(maxdav);
  DeviceVec<double> qnorm_d(nroots, prtlevel);
  HostVec<double>   qnorm_h(nroots);
  DeviceVec<int>    iroot_d(nroots, prtlevel);
  HostVec<int>      iroot_h(nroots);
  DeviceVec<int>    jrefconf_d(inidav, jrefconf, prtlevel);
  double            twall1, twall2, twall3;
  int               numdav = inidav;  // Current number of basis vectors.

  if (prtlevel >= 2)
  {
    // Print greeting message.
    printf("\n GPU version of the C++ implementation of the Davidson-Liu diagonalizer.\n\n");
    printf(" nci=%d, nroots=%d, inidav=%d, mindav=%d, maxdav=%d, kitdav=%d, qtol=%7.1le\n",
	   nci, nroots, inidav, mindav, maxdav, kitdav, qtol);
    double percent = 200.0 * (double)Ham_d.nnz / ((double)nci*(double)(nci+1));
    printf(" nnz=%ld (%4.2lf%)\n\n", Ham_d.nnz, percent);
    // fflush(stdout);
  }

  // Get initial wall clock time.
  gettime(0, 0, &twall1);
  twall2 = twall1;

  // Initial basis vectors.
  B_d.submat(nci, inidav).init(jrefconf_d);

  // Initialize projected Hamiltonian (G).
  P_d.submat(nci, numdav) =  Ham_d                   * B_d.submat(nci, numdav);
  G_d.subsym(numdav)      = ~B_d.submat(nci, numdav) * P_d.submat(nci, numdav);

  if (prtlevel >= 2)
  {
    // Print header for list of Davidson iterations.
    printf(" Iter.   Basis   To go   Worst      Eigenvalue           Norm      Wall clock   Total wall\n");
    //         1       30      5       5     3.123456789012345    0.1234E+12     1.23 s      123.45 s
    // fflush(stdout);
  }

  // Perform Davidson iterations.
  for (int iter=1;;)
  {
    // Temporary copy of G, will be destroyed by solving eigenvalue problem.
    A_h.submat(numdav, numdav).copyFromDevice(G_d);

    // Eigenvalues und eigenvectors of the submatrix.
    // A_h.submat(numdav, numdav).diag(eig_h);
    // A_d.submat(numdav, numdav).copyFromHost(A_h);
    // eig_d.copyFromHost(eig_h, numdav);

    // Unused columns of B and P will be used as scratch space.
    // If less than nroots columns are unused, a restart will be performed.
    if (numdav + nroots <= maxdav)
    {
      // Calculate nroots eigenvalues und eigenvectors of the submatrix.
      A_h.submat(numdav, numdav).eigen(U_h.submat(numdav, nroots), eig_h);
      A_d.submat(numdav, nroots).copyFromHost(U_h);
      eig_d.copyFromHost(eig_h, nroots);

      // Calculate Q matrix and column vector norms (appending Q to B).
      B_d.submat(nci, nroots, 0, numdav)  = P_d.submat(nci, numdav) * A_d.submat(numdav, nroots);
      U_d.submat(numdav, nroots).multByDiag(A_d.submat(numdav, nroots), eig_d);
      B_d.submat(nci, nroots, 0, numdav) -= B_d.submat(nci, numdav) * U_d.submat(numdav, nroots);
      B_d.submat(nci, nroots, 0, numdav).colnorms(qnorm_h);

      // Find roots that have not converged.
      int newdav = 0;  // Number of new basis vectors (one per unconverged root).

      for (int k=0; k<nroots; ++k)
        if (qnorm_h[k] > qtol)
          iroot_h[newdav++] = k;

      // Find root with the largest norm.
      int kbad = 0;  // Index of "worst" root (with the largest norm).

      for (int k=1; k<nroots; ++k)
        if (qnorm_h[k] > qnorm_h[kbad])
          kbad = k;

      // Get wall clock time.
      gettime(0, 0, &twall3);

      // Print information about iterations:
      if (prtlevel >= 2)
      {
        printf("%4d%9d%7d%8d%22.15lf%14.4le%9.2lf s%12.2lf s\n",
               iter, numdav, newdav, kbad+1, eig_h[kbad], qnorm_h[kbad],
               twall3-twall2, twall3-twall1);
        // fflush(stdout);
      }

      // Save current wall clock time:
      twall2 = twall3;

      // Terminate if all roots have converged.
      if (!newdav)
        break;

      // Error return if maximum number of iterations has been reached.
      if (iter >= kitdav)
      {
        printf(" Maximum number of Davidson iterations reached.\n");
        icall = -1;
        return;
      }

      // Calculate new basis vectors from Q matrix (appending Q' to P).
      iroot_d.copyFromHost(iroot_h, newdav);
      AddBasisVectors(P_d.submat(nci, newdav, 0, numdav), B_d.submat(nci, nroots, 0, numdav), Hii_d, eig_d, iroot_d, qtol);

      // Normalization factors of the column vectors of Q'.
      P_d.submat(nci, newdav, 0, numdav).colnorms(qnorm_h);
      for (int k=0; k<newdav; ++k)
        qnorm_h[k] = 1.0 / qnorm_h[k];

      // Normalize column vectors of Q' (in P).
      qnorm_d.copyFromHost(qnorm_h, newdav);
      P_d.submat(nci, newdav, 0, numdav).multByDiag(qnorm_d);

      // Schmidt orthogonalize (appending surviving vectors to B in the second step).
      Schmidt1(B_d.submat(nci, numdav),            P_d.submat(nci, newdav, 0, numdav), U_d.submat(numdav, newdav));
      Schmidt2(B_d.submat(nci, newdav, 0, numdav), P_d.submat(nci, newdav, 0, numdav), v_d, newdav, qtol);

      // Terminate if no new basis vectors have survived.
      if (newdav == 0)
        break;

      // Repeat orthogonalization (to remove numerical noise).
      Schmidt1(B_d.submat(nci, numdav),            B_d.submat(nci, newdav, 0, numdav), U_d.submat(numdav, newdav));
      Schmidt3(B_d.submat(nci, newdav, 0, numdav), v_d);

      // Update product matrix (P).
      P_d.submat(nci, newdav, 0, numdav)    = Ham_d * B_d.submat(nci, newdav, 0, numdav);

      // Update projected Hamiltonian (G) and counters.
      int numold  = numdav;
      numdav     += newdav;
      G_d.submat(newdav, numdav, numold, 0) = ~B_d.submat(nci, newdav, 0, numold) * P_d.submat(nci, numdav);
      ++iter;
    }
    else
    {
      // Perform a restart.

      // Calculate mindav eigenvalues und eigenvectors of the submatrix.
      A_h.submat(numdav, numdav).eigen(U_h.submat(numdav, mindav), eig_h);
      A_d.submat(numdav, mindav).copyFromHost(U_h);
      eig_d.copyFromHost(eig_h, mindav);

      // Collapse basis vectors (in B) into P and swap with B.
      P_d.submat(nci, mindav) = B_d.submat(nci, numdav) * A_d.submat(numdav, mindav);
      P_d.swap(B_d);
      numdav = mindav;

      // Schmidt orthogonalize (twice).
      Schmidt3(B_d.submat(nci, mindav), v_d);
      Schmidt3(B_d.submat(nci, mindav), v_d);

      // Calculate product matrix.
      P_d.submat(nci, numdav) = Ham_d * B_d.submat(nci, numdav);

      // Update projected Hamiltonian (G).
      G_d.subsym(numdav)      = ~B_d.submat(nci, numdav) * P_d.submat(nci, numdav);

      if (prtlevel >= 2)
        printf(" The Davidson algorithm has been restarted.\n");
    }
  }

  // Collapse basis vectors (in B) into P.
  P_d.submat(nci, nroots) = B_d.submat(nci, numdav) * A_d.submat(numdav, nroots);
  P_d.submat(nci, nroots).copyToHost(C, ldc);

  // Copy eigenvalues.
  for (int k=0; k<nroots; ++k)
    E[k] = eig_h[k];

  // Print statistics.
  long iend = Timing::GetTime(CLOCK_MONOTONIC);
  Timing::Print();
  printf("\n Overall wall clock time: %4.2lf s\n", 1e-9 * (double)(iend - ibegin));

  // Flush output to avoid mixing of Fortran and C output.
  fflush(stdout);
}


// Static members.
Counters Timing::ncalls;
Counters Timing::twall;

} // End of namespace.


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////   Interface functions.                                                                                                   ////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


extern "C"
{
  void gpudavliu_(double *Hii,
                  double *Hij,
                  int    *ifirst,
                  int    *icol,
                  int    &nci,
                  int    *jrefconf,
                  int    &inidav,
                  double *E,
                  double *C,
                  int    &ldc,
                  int    &nroots,
                  int    &mindav,
                  int    &maxdav,
                  int    &kitdav,
                  double &qtol,
                  int    &prtlevel,
                  int    &icall)
  {
    gpu::DavLiu(Hii, Hij, ifirst, icol, nci, jrefconf, inidav, E, C, ldc, nroots, mindav, maxdav, kitdav, qtol, prtlevel, icall);
  }
}
