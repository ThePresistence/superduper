
#include <hip/hip_runtime.h>
#include <cstring>


extern "C"
{
  void  chkgpu (int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu);
  void  chkgpu_(int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu);
  void _chkgpu (int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu);
  void _chkgpu_(int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu);
}


void chkgpu(int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu)
{
  // Query number of CUDA devices.
  if (hipGetDeviceCount(&numgpu) != hipSuccess)
  {
    numgpu = 0;
    return;
  }

  // Limit number of CUDA devices.
  if (numgpu > maxgpu)
    numgpu = maxgpu;

  // Initialize device names.
  memset(gpunam, ' ', 80 * numgpu);

  // Query properties of each CUDA device.
  for (int igpu=0; igpu<numgpu; ++igpu)
  {
    hipDeviceProp_t prop;

    if (hipGetDeviceProperties(&prop, igpu) == hipSuccess)
    {
      int len      = strlen(prop.name);
      memcpy(gpunam + 80 * igpu, prop.name, (len < 80) ? len : 80);
      nvcapa[igpu] = 10 * prop.major + prop.minor;
      mibglo[igpu] = static_cast<int>(static_cast<long long>(prop.totalGlobalMem + 524287LL) / 1048576LL);
    }
    else
    {
      // Querying device properties failed,
      // limit number of CUDA devices.
      numgpu = igpu;
      return;
    }
  }
}


void _chkgpu(int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu)
{
  chkgpu(numgpu, gpunam, nvcapa, mibglo, maxgpu);
}


void chkgpu_(int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu)
{
  chkgpu(numgpu, gpunam, nvcapa, mibglo, maxgpu);
}


void _chkgpu_(int &numgpu, char *gpunam, int *nvcapa, int *mibglo, int maxgpu)
{
  chkgpu(numgpu, gpunam, nvcapa, mibglo, maxgpu);
}
